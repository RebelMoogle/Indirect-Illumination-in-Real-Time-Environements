#include "hip/hip_runtime.h"
#include <optix_world.h>
#include <optixu/optixu_math_namespace.h>
#include "../OptixTypes.h"
#include "helpers.h"
//#include "random.h"

using namespace optix;

// Application input.
rtDeclareVariable(float3, Diffuse, , );
rtDeclareVariable(float3, LightPosition, , ) = {278.0f, 530.7f, 279.5f};
rtDeclareVariable(rtObject,      top_object, , );
rtDeclareVariable(float,         scene_epsilon, , ) = 0.01f;

// Intrinsic input.
rtDeclareVariable(float3,		hit_data,	  rtPayload, );

rtDeclareVariable(optix::Ray,	ray,          rtCurrentRay, );
rtDeclareVariable(float,		t_hit,        rtIntersectionDistance, );
rtDeclareVariable(uint2,		launch_index, rtLaunchIndex, );

// Input from Intersection program.
rtDeclareVariable(float3, shading_normal,		attribute shading_normal, );
rtDeclareVariable(float3, geometric_normal,		attribute geometric_normal, );


// Returns the hit color (does simple diffuse shading)
RT_PROGRAM void DirectDiffuse_closest_hit()
{
	float3 world_shading_normal   = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shading_normal ) );
	float3 world_geometric_normal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, geometric_normal ) );
	float3 ffnormal     = faceforward( world_shading_normal, -ray.direction, world_geometric_normal );
	float3 hit_point    = ray.origin + t_hit*ray.direction;

	float3 L = LightPosition - hit_point;
	float3 N = normalize(ffnormal);

	hit_data = dot(N,normalize(L)) * Diffuse;
}
/*
rtDeclareVariable(PhotonPRD,		hit_record,	  rtPayload, );
rtBuffer<PhotonRecord, 1>           PhotonMap;
//rtDeclareVariable(uint,		lin_launch_index, rtLaunchIndex, );

RT_PROGRAM void PhotonRay_closest_hit()
{
	// Check if this is a light source
	if (Diffuse.x == 0 && Diffuse.y == 0 && Diffuse.z == 0) return;

	float3 world_shading_normal   = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shading_normal ) );
	float3 world_geometric_normal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, geometric_normal ) );
	float3 ffnormal     = faceforward( world_shading_normal, -ray.direction, world_geometric_normal );

	float3 hit_point = ray.origin + t_hit*ray.direction;

	// ------	

//	uint2   seed     = photon_rnd_seeds[lin_launch_index];		
//	photon_rnd_seeds[lin_launch_index] = make_uint2(lcg2(seed.x), lcg2(seed.y));
//	float roulette = rnd(seed.x);
//	if( hit_record.ray_depth > 0 ) 	


	uint2&   seed     = hit_record.sample; //photon_rnd_seeds[lin_launch_index];		
//	photon_rnd_seeds[lin_launch_index] = make_uint2(lcg2(seed.x), lcg2(seed.y));
	float roulette = rnd_from_uint2(seed).x;
	
	PhotonRecord& rec = PhotonMap[hit_record.ray_index * DIFFUSE_PHOTONS_PER_RAY + hit_record.num_deposits];
	rec.position = hit_point;
	rec.normal = world_geometric_normal; //ffnormal;
	rec.ray_dir = ray.direction;
	rec.energy = hit_record.energy;
	hit_record.num_deposits++;

	if (roulette < 0.2) // Kill photon.		
		return;	

	hit_record.ray_depth++;
	if ( hit_record.num_deposits >= DIFFUSE_PHOTONS_PER_RAY || hit_record.ray_depth >= MAX_PHOTON_DEPTH)
		return;

	//hit_record.energy = Diffuse * hit_record.energy;
    hit_record.energy = Diffuse * hit_record.energy * 3.0f / (Diffuse.x + Diffuse.y + Diffuse.z); 
    float3 U, V, W;
    createONB(ffnormal, U, V, W);
	float3 new_ray_dir;
    sampleUnitHemisphere(rnd_from_uint2(seed), U, V, W, new_ray_dir);

	optix::Ray new_ray( hit_point, new_ray_dir, RayType_PhotonRay, scene_epsilon );
	rtTrace(top_object, new_ray, hit_record);
}
*/