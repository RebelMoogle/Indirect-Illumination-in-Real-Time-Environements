#include "hip/hip_runtime.h"

#include <optix_world.h>

using namespace optix;

// Application input.
rtDeclareVariable(float4, sphere, , );

// Intrinsic input.
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );

// Output to ClosestIntersection and AnyHit program.
rtDeclareVariable(float3, texcoord, attribute texcoord, ); 
rtDeclareVariable(float3, shading_normal, attribute shading_normal, ); 
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, ); 


template<bool use_robust_method>
__device__
void intersect_sphere(void)
{
  float3 center = make_float3(sphere);
  float3 O = ray.origin - center;
  float3 D = ray.direction;
  float radius = sphere.w;

  float b = dot(O, D);
  float c = dot(O, O)-radius*radius;
  float disc = b*b-c;
  if(disc > 0.0f){
    float sdisc = sqrtf(disc);
    float root1 = (-b - sdisc);

    bool do_refine = false;

    float root11 = 0.0f;

    if(use_robust_method && fabsf(root1) > 10.f * radius) {
      do_refine = true;
    }

    if(do_refine) {
      // refine root1
      float3 O1 = O + root1 * ray.direction;
      b = dot(O1, D);
      c = dot(O1, O1) - radius*radius;
      disc = b*b - c;

      if(disc > 0.0f) {
        sdisc = sqrtf(disc);
        root11 = (-b - sdisc);
      }
    }

    bool check_second = true;
    if( rtPotentialIntersection( root1 + root11 ) ) {
      shading_normal = geometric_normal = (O + (root1 + root11)*D)/radius;
      if(rtReportIntersection(0))
        check_second = false;
    } 
    if(check_second) {
      float root2 = (-b + sdisc) + (do_refine ? root1 : 0);
      if( rtPotentialIntersection( root2 ) ) {
        shading_normal = geometric_normal = (O + root2*D)/radius;
        rtReportIntersection(0);
      }
    }
  }
}


RT_PROGRAM void intersect(int primIdx)
{
  intersect_sphere<false>();
}

/*
RT_PROGRAM void robust_intersect(int primIdx)
{
  intersect_sphere<true>();
} */
/*
RT_PROGRAM void intersect(int primIdx)
{
	float3 center = make_float3(sphere);
	float3 O = ray.origin - center;
	float3 D = ray.direction;
	float r = sphere.w;

	//Compute A, B and C coefficients
	float a = dot(D, D);
	float b = 2 * dot(D, O);
	float c = dot(O, O) - (r * r);

    //Find discriminant
    float disc = b * b - 4 * a * c;
    
    // if discriminant is negative there are no real roots, so return 
    // false as ray misses sphere
    if (disc < 0)
        return;

    // compute q as described above
    float distSqrt = sqrtf(disc);
    float q;
    if (b < 0)
        q = (-b - distSqrt)/2.0;
    else
        q = (-b + distSqrt)/2.0;

    // compute t0 and t1
    float t0 = q / a;
    float t1 = c / q;

    // make sure t0 is smaller than t1
    if (t0 > t1)
    {
        // if t0 is bigger than t1 swap them around
        float temp = t0;
        t0 = t1;
        t1 = temp;
    }

    // if t1 is less than zero, the object is in the ray's negative direction
    // and consequently the ray misses the sphere
    if (t1 < 0)
        return;

    // if t0 is less than zero, the intersection point is at t1
    if (t0 < 0)
    {        
		if( rtPotentialIntersection( t1 ) ) {
			shading_normal = geometric_normal = (O + t1*D)/r;
			rtReportIntersection(0);
		}
    }
    // else the intersection point is at t0
    else
    {
        if( rtPotentialIntersection( t0 ) ) {
			shading_normal = geometric_normal = (O + t0*D)/r;
			rtReportIntersection(0);
		}
    }
} */


RT_PROGRAM void bounds (int, float result[6])
{
  const float3 cen = make_float3( sphere );
  const float3 rad = make_float3( sphere.w );

  optix::Aabb* aabb = (optix::Aabb*)result;
  
  if( rad.x > 0.0f  && !isinf(rad.x) ) {
    aabb->m_min = cen - rad;
    aabb->m_max = cen + rad;
  } else {
    aabb->invalidate();
  }
}