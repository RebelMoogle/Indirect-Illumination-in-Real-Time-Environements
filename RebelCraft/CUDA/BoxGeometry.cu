#include "hip/hip_runtime.h"

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <optixu/optixu_matrix_namespace.h>
#include <optixu/optixu_aabb_namespace.h>

using namespace optix;

// Application input.
rtDeclareVariable(float3, position, , );
rtDeclareVariable(float3, scale, , );

// Intrinsic input.
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );

// Output to ClosestIntersection and AnyHit program.
rtDeclareVariable(float3, texcoord, attribute texcoord, ); 
rtDeclareVariable(float3, shading_normal, attribute shading_normal, ); 
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, ); 


__device__ float3 boxnormal(float t)
{
  float3 t0 = (position - ray.origin)/ray.direction;
  float3 t1 = ((position + scale) - ray.origin)/ray.direction;
  float3 neg = make_float3(t==t0.x?1:0, t==t0.y?1:0, t==t0.z?1:0);
  float3 pos = make_float3(t==t1.x?1:0, t==t1.y?1:0, t==t1.z?1:0);
  return pos-neg;
}

RT_PROGRAM void intersect(int primIdx)
{
  float3 t0 = (position - ray.origin)/ray.direction;
  float3 t1 = ((position + scale) - ray.origin)/ray.direction;
  float3 near = fminf(t0, t1);
  float3 far = fmaxf(t0, t1);
  float tmin = fmaxf( near );
  float tmax = fminf( far );

  if(tmin <= tmax) {
    bool check_second = true;
    if( rtPotentialIntersection( tmin ) ) {
       texcoord = make_float3( 0.0f );
       shading_normal = geometric_normal = boxnormal( tmin );
       if(rtReportIntersection(0))
         check_second = false;
    } 
    if(check_second) {
      if( rtPotentialIntersection( tmax ) ) {
        texcoord = make_float3( 0.0f );
        shading_normal = geometric_normal = boxnormal( tmax );
        rtReportIntersection(0);
      }
    }
  }
}

RT_PROGRAM void bounds (int, float result[6])
{
  optix::Aabb* aabb = (optix::Aabb*)result;
  aabb->set(position, (position + scale) );
}