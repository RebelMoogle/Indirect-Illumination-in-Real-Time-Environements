#include "hip/hip_runtime.h"
#include <optix_world.h>
#include "helpers.h"
#include "..\\random.h"
#include "..\\OptixTypes.h"

using namespace optix;

// Application input.
rtDeclareVariable(float,			scene_epsilon, , );
rtDeclareVariable(rtObject,			top_object, , );
rtBuffer<uint2, 1>					randomSeeds;
rtBuffer<PhotonInitialBounce, 1>	photonInitialBuffer;
rtBuffer<PhotonRecord, 1>			photonResultBuffer;


// Intrinsic input.
rtDeclareVariable(uint, launch_index, rtLaunchIndex, );


RT_PROGRAM void photon_emitter()
{
	PhotonInitialBounce initialPhoton = photonInitialBuffer[launch_index];
	// Fetch random seeds.
	uint2  seed = make_uint2(initialPhoton.texCoord.x, initialPhoton.texCoord.y);//randomSeeds[launch_index];

	//	 Check if this photon has power
	//if ((initialPhoton.power.x + initialPhoton.power.y + initialPhoton.power.z)/3 < 0.01) return;

	float3 ray_origin = xyz(initialPhoton.position);
	float3 ray_direction = initialPhoton.direction;	
	
	optix::Ray ray(ray_origin, ray_direction, RayType_PhotonRay, scene_epsilon );

	// TODO: clear photon records before splatting!	
	// Initialize our photons
	/*for(unsigned int i = 0; i < DIFFUSE_PHOTONS_PER_RAY; ++i) {
		//prd.sample[i] = rnd_from_uint2(photon_rnd_seeds[launch_index]);
		photonResultBuffer[i+(launch_index) * DIFFUSE_PHOTONS_PER_RAY].incomingPower = make_float3(0.0f);
	}*/

	PhotonPRD prd;

	prd.power = xyz(initialPhoton.power);
	prd.sample = seed;
	prd.numDeposits = 0;
	prd.rayDepth = 0;
	//prd.pathDensity = initialPhoton.pathDensity;
	prd.etaIncoming = initialPhoton.texCoord.z;
	prd.travelDistance = initialPhoton.travelDistance;
	rtTrace( top_object, ray, prd );

	//// ### DEBUG ##

	//PhotonRecord dbgOutput;

	//dbgOutput.position = xyz(initialPhoton.position);
	//dbgOutput.normal = xyz(initialPhoton.normal);      // Pack this into 4 bytes
	//dbgOutput.incomingPower = xyz(initialPhoton.power);
	//dbgOutput.incomingDirection= initialPhoton.direction;
	//dbgOutput.travelDistance = initialPhoton.travelDistance;

	//photonResultBuffer[(launch_index) * DIFFUSE_PHOTONS_PER_RAY] = dbgOutput; 
}
