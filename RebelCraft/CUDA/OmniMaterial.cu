#include "hip/hip_runtime.h"
#include <optix_world.h>
#include <optixu/optixu_math_namespace.h>
#include "../OptixTypes.h"
#include "helpers.h"
#include "../random.h"

using namespace optix;

// Application input.
rtDeclareVariable(float3, Diffuse, , );
rtDeclareVariable(float4, Specular, , );
rtDeclareVariable(float4, Transmissive, , );
rtDeclareVariable(float3, LightPosition, , ) = {278.0f, 530.7f, 279.5f};
rtDeclareVariable(rtObject,      top_object, , );
rtDeclareVariable(float,         scene_epsilon, , ) = 0.01f;

// Intrinsic input.
rtDeclareVariable(float3,		hit_data,	  rtPayload, );

rtDeclareVariable(optix::Ray,	ray,          rtCurrentRay, );
rtDeclareVariable(float,		t_hit,        rtIntersectionDistance, );
rtDeclareVariable(uint2,		launch_index, rtLaunchIndex, );

// Input from Intersection program.
rtDeclareVariable(float3, shading_normal,		attribute shading_normal, );
rtDeclareVariable(float3, geometric_normal,		attribute geometric_normal, );


// Returns the hit color (does simple diffuse shading)
RT_PROGRAM void DirectDiffuse_closest_hit()
{
	float3 world_shading_normal   = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shading_normal ) );
	float3 world_geometric_normal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, geometric_normal ) );
	float3 ffnormal     = faceforward( world_shading_normal, -ray.direction, world_geometric_normal );
	float3 hit_point    = ray.origin + t_hit*ray.direction;

	float3 L = LightPosition - hit_point;
	float3 N = normalize(ffnormal);

	hit_data = dot(N,normalize(L)) * Diffuse;
}

rtDeclareVariable(PhotonPRD,		hit_prd,	  rtPayload, );
rtBuffer<PhotonRecord, 1>			photonResultBuffer;
rtDeclareVariable(uint,				lin_launch_index, rtLaunchIndex, );

__device__ bool Scatter(	uint2& seed, 
							float roulette,
							float3 normal, 
							float3 incomingDir, 
							float3 powerIncoming, float3 powerReflective, 
							float3 powerSpecular, float3 powerTransmissive,
							float etaMaterial, float specularExponent,
							float3& outgoingDir, float3& powerOutgoing, float& etaOutgoing) //, float& pathDensity)
{
	float ETA = hit_prd.etaIncoming;

	float powerReflectiveMean = (powerReflective.x + powerReflective.y + powerReflective.z) /3;

	float rouletteScatter = roulette - (powerReflectiveMean+0.3f);

	// diffuse scatter
	if( rouletteScatter <= 0.0f)
	{
		// TODO: multiply with material reflection data
		powerOutgoing = powerIncoming * powerReflective / powerReflectiveMean; /// * etaMaterial;
		//get the a diffuse reflection by using a normal hemisphere reflection ("random")

		// create tangents
		float3 U, V, W;
		createONB(normal, U, V, W);
		sampleUnitHemisphere(rnd_from_uint2(seed), U, V, W, outgoingDir);

		// This is a low-density path because
		// it scattered very diffusely
		//pathDensity *= powerReflectiveMean * 0.01;
		etaOutgoing = ETA; // lightETA is usually 1.0, is the incoming refractive index.

		//drop photon
		float3 hit_point = ray.origin + t_hit*ray.direction;
		PhotonRecord& rec = photonResultBuffer[lin_launch_index * DIFFUSE_PHOTONS_PER_RAY + hit_prd.numDeposits];
		rec.position = hit_point;
		rec.normal = normal; //world_geometric_normal; 
		rec.incomingDirection = ray.direction;
		rec.incomingPower = hit_prd.power;
		rec.travelDistance = hit_prd.travelDistance;
		//rec.pathDensity = pathDensity;
		hit_prd.numDeposits++;

		return true;
	}

	float powerSpecularMean = (powerSpecular.x + powerSpecular.y + powerSpecular.z) / 3;
	rouletteScatter = rouletteScatter - powerSpecularMean;
	if (rouletteScatter  <= 0.0) 
	{
		if (specularExponent < 1.0) 
		{
			// Glossy specular
			powerOutgoing = powerIncoming * powerSpecular / powerSpecularMean;
			outgoingDir = reflect(incomingDir, normal);//cosHemi(normal, random.g, random.b);

			float gloss = specularExponent * 127.0 + 1.0; //TODO: get rid of magic numbers
			//get the a specular reflection by using a squared hemisphere reflection around the reflected vector ("random")
			float3 U, V, W;
			createONB(normal, U, V, W);
			outgoingDir = sample_phong_lobe(rnd_from_uint2(seed), gloss, U, V, W);

			//outgoingDir = CosPowHemi(outgoingDir, gloss, random.g, random.b);

			// Medium density
			//pathDensity *= powerSpecularMean * 0.1;

			//drop photon
			float3 hit_point = ray.origin + t_hit*ray.direction;
			PhotonRecord& rec = photonResultBuffer[lin_launch_index * DIFFUSE_PHOTONS_PER_RAY + hit_prd.numDeposits];
			rec.position = hit_point;
			rec.normal = normal; //world_geometric_normal; 
			rec.incomingDirection = ray.direction;
			rec.incomingPower = hit_prd.power;
			//rec.pathDensity = pathDensity;
			rec.travelDistance = hit_prd.travelDistance;
			hit_prd.numDeposits++;

		} 
		else 
		{
			// Mirror specular
			powerOutgoing = powerIncoming * powerSpecular  / powerSpecularMean;

			// might be opposite?
			outgoingDir = reflect(incomingDir, normal);

			// High density
			//pathDensity *= powerSpecularMean;
		}
		etaOutgoing = ETA;

		return true;
	}

	//test, no transmissive yet!
	return false;

	float powerTransmitMean = (powerTransmissive.x + powerTransmissive.y + powerTransmissive.z) / 3;
	rouletteScatter -= powerTransmitMean;
	// Must be the case that this was transmissive
	if (rouletteScatter <= 0.0) 
	{         
		powerOutgoing = powerIncoming * powerTransmissive / powerTransmitMean;
		//might also be opposite
		 bool NoTotalReflection = refract(outgoingDir, incomingDir, normal, ETA / etaMaterial);

		// outgoingDir is zero on total internal refraction
		if (!NoTotalReflection) 
		{
			//pathDensity = powerTransmitMean;
			etaOutgoing = etaMaterial;
			return true;
		} 
		else 
		{
			powerOutgoing = powerIncoming * powerSpecular  / powerSpecularMean;
			outgoingDir = reflect(-incomingDir, normal);
			//pathDensity = powerSpecularMean;
			etaOutgoing = ETA;
			return true;
		}

	}

	return false;
}


RT_PROGRAM void PhotonRay_closest_hit()
{
	//	 Check if this is a light source
//	if (Diffuse.x == 0 && Diffuse.y == 0 && Diffuse.z == 0) return;

	float3 world_shading_normal   = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shading_normal ) );
	float3 world_geometric_normal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, geometric_normal ) );
	float3 ffnormal     = faceforward( world_shading_normal, -ray.direction, world_geometric_normal );

	float3 hit_point = ray.origin + t_hit*ray.direction;

	// get distance and add to traveldistance
	hit_prd.travelDistance += t_hit;


	// ------	scatter stuff
	float cosAngleIn = max(0.0f, dot(ray.direction, ffnormal));

	bool incident = cosAngleIn < 0;
	float refractionIndex = Transmissive.w;

	/*switch(hit_prd.numDeposits)
	{
	case 0:
		hit_prd.power = make_float3(1,0,0);
		break;
	case 1:
		hit_prd.power = make_float3(0,1,0);
		break;
	case 2:
		hit_prd.power = make_float3(0,0,1);
		break;
	case 3:
		hit_prd.power = make_float3(0,1,1);
		break;
	case 4:
		hit_prd.power = make_float3(1,1,0);
		break;
	case 5:
		hit_prd.power = make_float3(1,0,1);
		break;
	default:
		hit_prd.power = make_float3(1,1,1);
		break;

	}*/

	// instead of etaOutgoing ( should be carried along by photon) // instead of lightEta in scatter
	if(incident)
	{
		refractionIndex = refractionIndex / 1.0f;
	}
	else
	{
		refractionIndex = 1.0f / refractionIndex;
	}
	float3 powerSpecular = fresnel_schlick(cosAngleIn, 5.0f, xyz(Specular), make_float3(1.0f));
	float3 powerTransmissive = xyz(Transmissive);

	uint2&   seed     = hit_prd.sample;
	float roulette = rnd_from_uint2(seed).x;
	
	float3 powerTotal = Diffuse + powerSpecular + powerTransmissive;

	/*if(roulette <= (powerTotal.x + powerTotal.y + powerTotal.z)/3)
	{
		// drop photon
		PhotonRecord& rec = photonResultBuffer[lin_launch_index * DIFFUSE_PHOTONS_PER_RAY + hit_prd.numDeposits];
		rec.position = hit_point;
		rec.normal = world_geometric_normal; //ffnormal;
		rec.incomingDirection = ray.direction;
		rec.incomingPower = hit_prd.power;
		//rec.pathDensity = hit_prd.pathDensity;
		rec.travelDistance = hit_prd.travelDistance;
		hit_prd.numDeposits++;

		return; //Kill it
	}*/

	float3 outgoingDir = make_float3(0,0,0);
	float3 outgoingPower = make_float3(0,0,0);
	float  outgoingETA = 0.0f;
	//float  pathDensity;
	if(!Scatter(seed, roulette, ffnormal, ray.direction, hit_prd.power, Diffuse, powerSpecular, powerTransmissive, Transmissive.w, Specular.w, outgoingDir, outgoingPower, outgoingETA)) //, pathDensity))
	{
		//default diffuse on kill
		// drop photon
			PhotonRecord& rec = photonResultBuffer[lin_launch_index * DIFFUSE_PHOTONS_PER_RAY + hit_prd.numDeposits];
			rec.position = hit_point;
			rec.normal = world_geometric_normal; //ffnormal;
			rec.incomingDirection = ray.direction;
			rec.incomingPower = hit_prd.power;
			//rec.pathDensity = hit_prd.pathDensity;
			rec.travelDistance = hit_prd.travelDistance;
			hit_prd.numDeposits++;;

			return;
	}
	
	hit_prd.rayDepth++;
	if ( hit_prd.numDeposits >= DIFFUSE_PHOTONS_PER_RAY) //|| hit_prd.rayDepth >= MAX_PHOTON_DEPTH || outgoingETA == 0.0f)
		return;

	//hit_prd.energy = Diffuse * hit_prd.energy;
    hit_prd.power = outgoingPower; //may lose saturation 
	//hit_prd.pathDensity = pathDensity;
	hit_prd.etaIncoming = outgoingETA;

	optix::Ray new_ray( hit_point, outgoingDir, RayType_PhotonRay, scene_epsilon );
	rtTrace(top_object, new_ray, hit_prd);
}