#include "hip/hip_runtime.h"
#include <optix_world.h>
#include "helpers.h"
#include "packing.h"

using namespace optix;

rtDeclareVariable(float3,        eye, , );
rtDeclareVariable(float3,        U, , );
rtDeclareVariable(float3,        V, , );
rtDeclareVariable(float3,        W, , );
rtDeclareVariable(float,         scene_epsilon, , );
rtDeclareVariable(rtObject,      top_object, , );
rtDeclareVariable(unsigned int,  ray_type, , );
rtDeclareVariable(unsigned int,  Width, , );
rtDeclareVariable(unsigned int,  Height, , );

rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
rtDeclareVariable(uint2, launch_dim,   rtLaunchDim, );

// Output buffer.
rtBuffer<uint, 2>			raycast_buffer;

RT_PROGRAM void optix_viewer_camera()
{
  float2 d = make_float2(launch_index.x + 1, launch_index.y + 1) / make_float2(launch_dim.x + 1, launch_dim.y + 1) * 2.f - 1.f;
  float3 ray_origin = eye;
  float3 ray_direction = normalize(d.x*U + d.y*V + W);
  
  optix::Ray ray = optix::make_Ray(ray_origin, ray_direction, ray_type, scene_epsilon, RT_DEFAULT_MAX);

  float3 prd = make_float3(0,0,0);
  rtTrace(top_object, ray, prd);

  //raycast_buffer[launch_index] = float4_to_R8G8B8A8_UNORM(make_float4(1,1,0,1));
  raycast_buffer[launch_index] = float4_to_R8G8B8A8_UNORM(make_float4(prd, 1));
}
